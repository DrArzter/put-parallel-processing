#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

__global__ void count_points(int *count, int N, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        double x = hiprand_uniform(&state);
        double y = hiprand_uniform(&state);
        if (x * x + y * y <= 1.0) {
            atomicAdd(count, 1);
        }
    }
}

int main() {
    const int N = 1000000;
    int count = 0;
    int *d_count;
    hipMalloc(&d_count, sizeof(int));
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    count_points<<<blocksPerGrid, threadsPerBlock>>>(d_count, N, time(0));

    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_count);

    double pi = 4.0 * count / N;
    std::cout << "Approximate value of pi: " << pi << std::endl;
    return 0;
}
